#include "hip/hip_runtime.h"
/* Blur filter. Device code. */

#ifndef _BLUR_FILTER_KERNEL_H_
#define _BLUR_FILTER_KERNEL_H_

#include "blur_filter.h"

__global__ void blur_filter_kernel (const float *in, float *out, int size)
{
	int row = blockIdx.y * blockDim.y + threadIdx.y;
	int col = blockIdx.x * blockDim.x + threadIdx.x;

	if ((row < size) && (col < size)) {
		float blur_value = 0.0;
		int num_neighbors = 0;

		for (int i = -BLUR_SIZE; i < (BLUR_SIZE + 1); i++) {
			for (int j = -BLUR_SIZE; j < (BLUR_SIZE + 1); j++) {
				int curr_row = row + i;
				int curr_col = col + j;

				if ((curr_row > -1) && (curr_row < size)  && (curr_col > -1) && (curr_col < size)) {
					blur_value += in[curr_row * size + curr_col];
					num_neighbors += 1;
				}
			}
		}

		out[row * size + col] = blur_value / num_neighbors;
	}
}

#endif /* _BLUR_FILTER_KERNEL_H_ */
