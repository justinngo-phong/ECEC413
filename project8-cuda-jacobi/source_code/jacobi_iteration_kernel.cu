#include "hip/hip_runtime.h"
#include "jacobi_iteration.h"

/* Device kernels to solve the Jacobi iterations */


__global__ void jacobi_iteration_kernel_naive(float *A, float *B, float *x, float *new_x, double *ssd)
{
	extern __shared__ double s_ssd[];

	int i = blockIdx.x * blockDim.x + threadIdx.x;
	int n = MATRIX_SIZE;

	double sum = 0.0;
	for (int j = 0; j < n; j++) {
		if (i != j)
			sum += A[i * n + j] * x[j];
	}

	new_x[i] = (B[i] - sum) / A[i * n + i];

	double diff = new_x[i] - x[i];
	s_ssd[threadIdx.x] = diff * diff;

	__syncthreads();

	/* Parallel reduction within a block */
	for (unsigned int stride = blockDim.x >> 1; stride > 0; stride >>= 1) {
		if (threadIdx.x < stride)
			s_ssd[threadIdx.x] += s_ssd[threadIdx.x + stride];
		__syncthreads();
	}

	/* Store ssd back to global memory */
	if (threadIdx.x == 0)
		atomicAdd(ssd, s_ssd[0]);
}

__global__ void transpose_matrix_kernel(float *A, float *A_col)
{
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	int j = blockIdx.y * blockDim.y + threadIdx.y;
	int n = MATRIX_SIZE;

	A_col[i * n + j] = A[j * n + i];
}

__global__ void jacobi_iteration_kernel_optimized(float *A_col, float *B, float *x, float *new_x, double *ssd)
{
	extern __shared__ double s_ssd[];

	int i = blockIdx.x * blockDim.x + threadIdx.x;
	int n = MATRIX_SIZE;

	double sum = 0.0;
	for (int j = 0; j < n; j++) {
		if (i != j)
			sum += A_col[j * n + i] * x[j];  // Access A_col in column-major order
	}
	__syncthreads();

	new_x[i] = (B[i] - sum) / A_col[i * n + i];  // Access A_col in column-major order

	double diff = new_x[i] - x[i];
	s_ssd[threadIdx.x] = diff * diff;

	__syncthreads();

	/* Parallel reduction within a block */
	for (unsigned int stride = blockDim.x >> 1; stride > 0; stride >>= 1) {
		if (threadIdx.x < stride)
			s_ssd[threadIdx.x] += s_ssd[threadIdx.x + stride];
		__syncthreads();
	}

	/* Store ssd back to global memory */
	if (threadIdx.x == 0)
		atomicAdd(ssd, s_ssd[0]);
}

